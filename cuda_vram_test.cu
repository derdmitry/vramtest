#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

const int bytesPerGB = 1024 * 1024 * 1024; // 1 GB


__global__ void init_random(float *data, size_t num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        hiprandState state;
        hiprand_init(clock64(), idx, 0, &state);
        data[idx] = hiprand_uniform(&state) * 100.0f;
    }
}

int main() {
    int device_count = 0;
    hipError_t error = hipGetDeviceCount(&device_count);
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    std::cout << "Find " << device_count << " CUDA devices:" << std::endl;

    for (int i = 0; i < device_count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "\nDevice " << i << ": " << prop.name << std::endl;
        std::cout << "  Total VRAM: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;

        size_t totalAllocated = 0;
        std::vector<float*> pointers;

        while (totalAllocated + bytesPerGB <= prop.totalGlobalMem) {
            float* d_data;
            size_t num_elements = bytesPerGB / sizeof(float);

            error = hipMalloc(&d_data, bytesPerGB);
            if (error != hipSuccess) {
                std::cerr << "Memory allocation error: " << hipGetErrorString(error) << std::endl;
                break; // 
            }

            pointers.push_back(d_data);
            totalAllocated += bytesPerGB;

            std::cout << "Allocated " << totalAllocated / (1024 * 1024) << " MB" << std::endl;

            // 
            int threadsPerBlock = 256;
            int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
            init_random<<<blocksPerGrid, threadsPerBlock>>>(d_data, num_elements);
            hipDeviceSynchronize();
        }

        // 
        for (auto ptr : pointers) {
            hipFree(ptr);
        }
    }

    return 0;
}
